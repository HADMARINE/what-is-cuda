#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define ITER 100000
#define THREAD_PER_BLOCK 10
#define PI 3.1415926535
#define RAD(X) X *(PI / 180.0)

__global__ void calculator_kernel(float *sin_arr, float *cos_arr, float *tan_arr, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {
        float rad = RAD(idx);
        sin_arr[idx] = sinf(rad);
        cos_arr[idx] = cosf(rad);
        tan_arr[idx] = tanf(rad);
    }
}

int main()
{
    hipProfilerStart();

    float *sin_arr, *cos_arr, *tan_arr;

    hipMallocManaged((void **)&sin_arr, sizeof(float) * ITER);
    hipMallocManaged((void **)&cos_arr, sizeof(float) * ITER);
    hipMallocManaged((void **)&tan_arr, sizeof(float) * ITER);

    calculator_kernel<<<ITER / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(sin_arr, cos_arr, tan_arr, ITER);
    hipDeviceSynchronize();

    for (int i = 0; i < ITER; i++)
    {
        printf("sin (%d) = %f cos (%d) = %f tan (%d) = %f\n", i, sin_arr[i], i, cos_arr[i], i, tan_arr[i]);
    }

    hipFree(sin_arr);
    hipFree(cos_arr);
    hipFree(tan_arr);

    hipProfilerStop();

    return 0;
}