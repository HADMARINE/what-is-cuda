#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

__global__ void loop_gpu()
{
    printf("GPU Loop, NUM : %d\n", threadIdx.x);
}

void loop_cpu(int N)
{
    for (int i = 0; i < N; i++)
    {
        printf("CPU Loop, NUM : %d\n", i);
    }
}

int main()
{
    hipProfilerStart();

    int loop_count = 10;
    loop_cpu(loop_count);
    loop_gpu<<<1, loop_count>>>();
    hipDeviceSynchronize();

    hipProfilerStop();

    return 0;
}