#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void some_kernel(int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N)
    {
        printf("this is valid.")
    }
}

int main()
{
}